#include "hip/hip_runtime.h"
#include "my_cuda.cuh"

using namespace std;

static float(*device1_float_out)[4] = NULL, (*device2_float_out)[4] = NULL, (*device3_float_out)[4] = NULL, (*device4_float_out)[4] = NULL;
static float(*Amp_real_out)[4] = NULL, (*Phi_imag_out)[4] = NULL;
static hipfftComplex(*U0_device_out)[4] = NULL;
hipStream_t stream[4];

#define PI 3.14159265358979323846

//std::mutex mtx_1;

static void Check(hipError_t status)
{
	if (status != hipSuccess)
	{
		cout << "�к�:" << __LINE__ << endl;
		cout << "����:" << hipGetErrorString(status) << endl;
	}
}

__global__ void Mat2complex_kernel(float* src_1, float* src_2, hipfftComplex* U0, bool phase_mode, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N) {
		if (phase_mode) {
			U0[idx].x = src_1[idx] * cos(src_2[idx]);
			U0[idx].y = src_1[idx] * sin(src_2[idx]);
		}
		else {
			//�о���ط�û��ҪCUDA���٣�PC����Ҳû�õ�
			U0[idx].x = src_1[idx];
			U0[idx].y = src_2[idx];
		}
	}
}

__global__ void Mat2complex_kernel(float* src_2, hipfftComplex* U0, float a, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N) {
		U0[idx].x = a * cos(src_2[idx]);
		U0[idx].y = a * sin(src_2[idx]);
	}
}

__global__ void Complex2mat_kernel(hipfftComplex* U0, float* dst_1, float* dst_2, bool phase_mode, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N) {
		if (phase_mode) {
			dst_1[idx] = sqrt(U0[idx].x * U0[idx].x + U0[idx].y * U0[idx].y);
			dst_2[idx] = atan2(U0[idx].y, U0[idx].x);
		}
		else {
			//��������ܼ����𣿲��ᷴ�������ٶ���
			dst_1[idx] = U0[idx].x;
			dst_2[idx] = U0[idx].y;
		}
	}
}

__global__ void grid_creat_kernel(float* dst, int nH, int nW, float delta_x, float delta_y, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N) {
		dst[idx] = sqrt((-nW / 2.0f * delta_x + idx % nW * delta_x)\
					  * (-nW / 2.0f * delta_x + idx % nW * delta_x)\
					  + (-nH / 2.0f * delta_y + idx / nW * delta_y)\
					  * (-nH / 2.0f * delta_y + idx / nW * delta_y));
	}
}

__global__ void CUDA_mul_kernel_2(float* src1, float* src2, float* src3, float* src4, float* dst1, float* dst2, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N) {
		dst1[idx] = src1[idx] * src3[idx] - src2[idx] * src4[idx];
		dst2[idx] = src1[idx] * src4[idx] + src2[idx] * src3[idx];
	}
}

__global__ void CUDA_mul_kernel(float* src1, float* src2, float* dst, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N) {
		dst[idx] = src1[idx] * src2[idx];
	}
}

//FFT���任�����ڹ淶���ĺ���
__global__ void normalizing(hipfftComplex* data, int data_len)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	data[idx].x /= data_len;
	data[idx].y /= data_len;
}

__global__ void filter_creat_kernel(float* src, float* dst1, float* dst2, float Max_frequency, float Filter, float Denoise_Radius, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N) {
		dst1[idx] = (float)(src[idx] < Max_frequency * Filter);
		dst2[idx] = (float)(src[idx] < Denoise_Radius);
	}
}
//���� ʵ��fftshift
__global__ void fftshift_step1_kernel(float* src, float* dst1, float* dst2, float* dst3, float* dst4, int nH, int nW, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N) {
		if ((idx % nW < nW / 2) && (idx / nW < nH / 2))
		{
			dst1[idx / nW * (nW / 2) + idx % nW] = src[idx];
		}
		else if ((idx % nW >= nW / 2) && (idx / nW < nH / 2))
		{
			dst2[idx / nW * (nW / 2) + (idx - nW / 2) % nW] = src[idx];
		}
		else if ((idx % nW < nW / 2) && (idx / nW >= nH / 2))
		{
			dst3[idx / nW * (nW / 2) - nH / 2 * nW / 2 + idx % nW] = src[idx];
		}
		else
		{
			dst4[idx / nW * (nW / 2) - nH / 2 * nW / 2 + (idx - nW / 2) % nW] = src[idx];
		}
	}
}

__global__ void fftshift_step2_kernel(float* tmp1, float* tmp2, float* tmp3, float* tmp4, float* Phi, int nH, int nW, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N) {
		if ((idx % nW < nW / 2) && (idx / nW < nH / 2))
		{
			Phi[idx] = tmp4[idx / nW * (nW / 2) + idx % nW];
		}
		else if ((idx % nW >= nW / 2) && (idx / nW < nH / 2))
		{
			Phi[idx] = tmp3[idx / nW * (nW / 2) + (idx - nW / 2) % nW];
		}
		else if ((idx % nW < nW / 2) && (idx / nW >= nH / 2))
		{
			Phi[idx] = tmp2[idx / nW * (nW / 2) - nH / 2 * nW / 2 + idx % nW];
		}
		else
		{
			Phi[idx] = tmp1[idx / nW * (nW / 2) - nH / 2 * nW / 2 + (idx - nW / 2) % nW];
		}
	}
}

__global__ void Phase_delay(float* data, bool flag, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N) {
		if (flag){
			data[idx] = data[idx] * PI / 2;
		}
		else {
			data[idx] = -data[idx] * PI / 2;
		}
	}
}

__global__ void rect_kernel(float* src, float* dst, int nH, int nW, int x, int y, int high, int width, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N && idx % nW>y - 1 && idx % nW<y + width && idx / nW>x - 1 && idx / nW < x + high) {
		dst[(idx / nW - x) * width + (idx - idx / nW * nW - y)] = src[idx];
	}
}

void CUDA_Init_ALL(int nH, int nW)
{
	size_t a, b;
	Check(hipMalloc((void**)&device1_float_out, 4 * nH * nW * sizeof(float)));
	Check(hipMalloc((void**)&device2_float_out, 4 * nH * nW * sizeof(float)));
	Check(hipMalloc((void**)&device3_float_out, 4 * nH * nW * sizeof(float)));
	Check(hipMalloc((void**)&device4_float_out, 4 * nH * nW * sizeof(float)));
	Check(hipMalloc((void**)&Amp_real_out, 4 * nH * nW * sizeof(float)));
	Check(hipMalloc((void**)&Phi_imag_out, 4 * nH * nW * sizeof(float)));
	Check(hipMalloc((void**)&U0_device_out, 4 * nH * nW * sizeof(hipfftComplex)));
	
	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);
	hipStreamCreate(&stream[2]);
	hipStreamCreate(&stream[3]);

}
//
void CUDA_Free_ALL()
{
	Check(hipFree(device1_float_out));
	Check(hipFree(device2_float_out));
	Check(hipFree(device3_float_out));
	Check(hipFree(device4_float_out));
	Check(hipFree(Amp_real_out));
	Check(hipFree(Phi_imag_out));
	Check(hipFree(U0_device_out));
}

//�������񲢼���ƽ����
void CUDA_grid_creat_now(cv::Mat* dst, int nH, int nW, float delta_x, float delta_y, int i)
{
	printf("abc = %d", i);
	float* device1_float = device1_float_out[i];
	cv::Mat rho = cv::Mat::ones(nH, nW, CV_32FC1);
	dim3 grid((nH * nW + 1024 - 1) / 1024);//(1280*960+1024-1)/1024
	dim3 block(1024);//1024
	grid_creat_kernel << <grid, block >> > (device1_float, nH, nW, delta_x, delta_y, nH * nW);
	hipDeviceSynchronize();
	Check(hipMemcpy(rho.data, (uchar*)device1_float, nH * nW * sizeof(float), hipMemcpyDeviceToHost));
	*dst = rho;
	device1_float = NULL;
	//hipStreamSynchronize(0);
}

void CUDA_circ_creat(cv::Mat src, cv::Mat& dst1, cv::Mat& dst2, float Max_frequency, float Filter, int i)
{
	int nH = src.rows;
	int nW = src.cols;
	int Nt = nH * nW;
	/*�ݶ�����*/
	int PS_Gas_Radius = 15;// PS_Gas_RadiusΪ��˹�˲��뾶���ø�˹�˲�ʵ����ֺ���ƣ�
	int PS_Gas_var = 60;// PS_Gas_varΪ��˹�˲�����ø�˹�˲�ʵ����ֺ���ƣ�
	float Denoise_Radius = (1.0f / 3.0f);;// Denoise_Radius��ȥ��뾶
	int G_R = 31; // gaussian�˲��뾶
	int G_V = 300; // gaussian�˲�����

	float* device1_float = device1_float_out[i];
	float* device2_float = device2_float_out[i];
	float* device3_float = device3_float_out[i];

	cv::Mat weight = cv::Mat::zeros(nH, nW, CV_32FC1);
	cv::Mat filter2 = cv::Mat::zeros(nH, nW, CV_32FC1);

	Check(hipMemcpy(device3_float, src.data, nH * nW * sizeof(float), hipMemcpyHostToDevice));
	dim3 grid((nH * nW + 1024 - 1) / 1024);//(1280*960+1024-1)/1024
	dim3 block(1024);//1024
	filter_creat_kernel << <grid, block >> > (device3_float, device1_float, device2_float, Max_frequency, Filter, Denoise_Radius, Nt);
	hipDeviceSynchronize();
	Check(hipMemcpy(weight.data, (uchar*)device1_float, nH * nW * sizeof(float), hipMemcpyDeviceToHost));
	Check(hipMemcpy(filter2.data, (uchar*)device2_float, nH * nW * sizeof(float), hipMemcpyDeviceToHost));
	dst1 = weight;
	dst2 = filter2;
	//hipStreamSynchronize(0);
}

void CUDA_filter_creat_now(cv::Mat src1, cv::Mat src2, cv::Mat& dst1, cv::Mat& dst2, bool flag, int i)
{
	int nH = src2.rows;
	int nW = src2.cols;
	int Nt = nH * nW;
	
	float* device1_float = device1_float_out[i];
	float* device2_float = device2_float_out[i];
	float* device3_float = device3_float_out[i];
	float* device4_float = device4_float_out[i];
	float* Amp_real = Amp_real_out[i];
	float* Phi_imag = Phi_imag_out[i];
	hipfftComplex* U0_device = U0_device_out[i];

	cv::Mat real_out = cv::Mat::zeros(nH, nW, CV_32FC1);
	cv::Mat imag_out = cv::Mat::zeros(nH, nW, CV_32FC1);
	Check(hipMemcpy(device1_float, src1.data, nH * nW * sizeof(float), hipMemcpyHostToDevice));
	Check(hipMemcpy(device2_float, src2.data, nH * nW * sizeof(float), hipMemcpyHostToDevice));

	dim3 grid((nH * nW + 1024 - 1) / 1024);//(1280*960+1024-1)/1024
	dim3 block(1024);//1024
	Phase_delay << <grid, block >> > (device1_float, flag, Nt);
	Mat2complex_kernel << <grid, block >> > (device1_float, U0_device, 1.0f, Nt);
	Complex2mat_kernel << <grid, block >> > (U0_device, Amp_real, Phi_imag, false, Nt);
	CUDA_mul_kernel << <grid, block >> > (Amp_real, device2_float, Amp_real, Nt);
	CUDA_mul_kernel << <grid, block >> > (Phi_imag, device2_float, Phi_imag, Nt);
	
	//fftshift real
	fftshift_step1_kernel << <grid, block >> > (Amp_real, device1_float, device2_float, device3_float, device4_float, nH, nW, Nt);
	fftshift_step2_kernel << <grid, block >> > (device1_float, device2_float, device3_float, device4_float, Amp_real, nH, nW, Nt);
	Check(hipMemcpy(real_out.data, (uchar*)Amp_real, nH * nW * sizeof(float), hipMemcpyDeviceToHost));
	dst1 = real_out;
	//fftshift imag
	fftshift_step1_kernel << <grid, block >> > (Phi_imag, device1_float, device2_float, device3_float, device4_float, nH, nW, Nt);
	fftshift_step2_kernel << <grid, block >> > (device1_float, device2_float, device3_float, device4_float, Phi_imag, nH, nW, Nt);
	Check(hipMemcpy(imag_out.data, (uchar*)Phi_imag, nH * nW * sizeof(float), hipMemcpyDeviceToHost));
	dst2 = imag_out;
	//hipStreamSynchronize(0);
}

void CUDA_ALL_calculate(cv::Mat Phi, cv::Mat real_filter, cv::Mat imag_filter, cv::Mat* Ipc, int nH_extend, int nW_extend, int i)
{
	int nH = real_filter.rows;
	int nW = real_filter.cols;
	int nH_old = nH - nH_extend * 2;
	int nW_old = nW - nW_extend * 2;
	int Nt = nH * nW;
	float a = 0.7f;

	float* device1_float = device1_float_out[i];
	float* device2_float = device2_float_out[i];
	float* device3_float = device3_float_out[i];
	float* device4_float = device4_float_out[i];
	float* Amp_real = Amp_real_out[i];
	float* Phi_imag = Phi_imag_out[i];
	hipfftComplex* U0_device = U0_device_out[i];

	hipfftHandle cufftForwrdHandle, cufftInverseHandle;

	hipfftPlan2d(&cufftForwrdHandle, nH, nW, HIPFFT_C2C);
	hipfftPlan2d(&cufftInverseHandle, nH, nW, HIPFFT_C2C);

	hipfftSetStream(cufftForwrdHandle, stream[i]);
	hipfftSetStream(cufftInverseHandle, stream[i]);

	cv::Mat Ipc_out = cv::Mat::zeros(nH_old, nW_old, CV_32FC1);

	Check(hipMemcpyAsync(Phi_imag, Phi.data, nH * nW * sizeof(float), hipMemcpyHostToDevice, stream[i]));
	Check(hipMemcpyAsync(Amp_real, real_filter.data, nH * nW * sizeof(float), hipMemcpyHostToDevice, stream[i]));

	dim3 grid((nH * nW + 1024 - 1) / 1024);//(1280*960+1024-1)/1024
	dim3 block(1024);//1024
	Mat2complex_kernel << <grid, block, 0, stream[i] >> > (Phi_imag, U0_device, a, Nt);
	//ִ��fft���任
	hipfftExecC2C(cufftForwrdHandle, U0_device, U0_device, HIPFFT_FORWARD);
	
	Complex2mat_kernel << <grid, block, 0, stream[i] >> > (U0_device, device1_float, device2_float, false, Nt);

	Check(hipMemcpyAsync(Phi_imag, imag_filter.data, nH * nW * sizeof(float), hipMemcpyHostToDevice, stream[i]));
	
	CUDA_mul_kernel_2 << <grid, block, 0, stream[i] >> > (Amp_real, Phi_imag, device1_float, device2_float, device3_float, device4_float, Nt);

	Mat2complex_kernel << <grid, block, 0, stream[i] >> > (device3_float, device4_float, U0_device, false, Nt);

	hipfftExecC2C(cufftForwrdHandle, U0_device, U0_device, HIPFFT_BACKWARD);
	
	normalizing << <grid, block, 0, stream[i] >> > (U0_device, Nt);
	Complex2mat_kernel << <grid, block, 0, stream[i] >> > (U0_device, Amp_real, Phi_imag, true, Nt);
	
	CUDA_mul_kernel << <grid, block, 0, stream[i] >> > (Amp_real, Amp_real, device1_float, Nt);

	rect_kernel << <grid, block, 0, stream[i] >> > (device1_float, device3_float, nH, nW, nH_extend, nW_extend, nH_old, nW_old, Nt);

	Check(hipMemcpyAsync(Ipc_out.data, (uchar*)device3_float, nH_old * nW_old * sizeof(float), hipMemcpyDeviceToHost, stream[i]));
	*Ipc = Ipc_out;
	
}

//void CUDA_gaussianBlur_gpu(cv::Mat & src, cv::Mat & dst, int Gas_Radius, int Gas_var)
//{
//	cv::cuda::GpuMat src_gpu, dst_gpu;
//
//	src_gpu.upload(src);
//
//	cv::Ptr<cv::cuda::Filter> filter;
//	filter = cv::cuda::createGaussianFilter(CV_32FC1, CV_32FC1, cv::Size(Gas_Radius, Gas_Radius), Gas_var, Gas_var);
//	filter->apply(src_gpu, dst_gpu);
//	dst_gpu.download(dst);
//}